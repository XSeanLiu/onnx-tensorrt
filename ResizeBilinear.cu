#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <cassert>
#include <algorithm>
#include "ResizeBilinear.hpp"

// TODO: Move this to a common header
inline bool is_CHW(nvinfer1::Dims const& dims) {
  return (dims.nbDims == 3 &&
          dims.type[0] == nvinfer1::DimensionType::kCHANNEL &&
          dims.type[1] == nvinfer1::DimensionType::kSPATIAL &&
          dims.type[2] == nvinfer1::DimensionType::kSPATIAL);
}

nvinfer1::Dims ResizeBilinearPlugin::getOutputDimensions(int index,
                                                        const nvinfer1::Dims *inputDims,
                                                        int nbInputs) {
  assert(nbInputs == 1);
  nvinfer1::Dims const& input = inputDims[0];
  assert(is_CHW(input));
  assert(_ndims == 2);
  assert(index == 0);
  nvinfer1::Dims output;
  output.nbDims = input.nbDims;
  int s = 0;
  for( int d=0; d<input.nbDims; ++d ) {
    output.type[d] = input.type[d];
    if( input.type[d] == nvinfer1::DimensionType::kSPATIAL ) {
      output.d[d] = int(input.d[d] * _scale[s++]);
    } else {
      output.d[d] = input.d[d];
    }
  }
  return output;
}

int ResizeBilinearPlugin::initialize() {
  _output_dims = this->getOutputDimensions(0, &this->getInputDims(0), 1);
  assert(is_CHW(this->getInputDims(0)));
  assert(is_CHW(_output_dims));
  assert(_ndims == 2);
  return 0;
}

__device__
void area_pixel_compute_source_index(float &rc,
                                     float scale,
                                     int dst_index,
                                     bool align_corners,
                                     bool cubic = false)
{
    if (align_corners)
    {
        rc = scale * dst_index;
        return;
    }
    else
    {
        float src_idx = scale * (dst_index + 0.5) - 0.5;
        rc = (!cubic && src_idx < 0) ? float(0.0) : src_idx;
        return;
    }
}

template <typename Data>
__global__
void resize_bilinear_kernel_2d(int n,
                               int batchsize,
                               int channels,
                               int height1,
                               int width1,
                               int height2,
                               int width2,
                               float rheight,
                               float rwidth,
                               bool align_corners,
                               Data const* idata,
                               Data*       odata) 
{
    const int in_batchsize_stride = channels * height1 * width1;
    const int in_channels_stride = height1 * width1;
    const int out_batchsize_stride = channels * height2 * width2;
    const int out_channels_stride = height2 * width2;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        const int w2 = index % width2;
        const int h2 = index / width2;
        if (height1 == height2 && width1 == width2)
        {
            const int h1 = h2;
            const int w1 = w2;
            for (int n = 0; n < batchsize; n++)
            {
                for (int c = 0; c < channels; c++)
                {
                    odata[n * out_batchsize_stride + c * out_channels_stride + h2 * width2 + w2]
                        = idata[n * in_batchsize_stride + c * in_channels_stride + h1 * width1 + w1];
                }
            }
            return;
        }
        //
        float h1r;
        area_pixel_compute_source_index(h1r, rheight, h2, align_corners, /*cubic=*/false);
        const int h1 = h1r;
        const int h1p = (h1 < height1 - 1) ? 1 : 0;
        const float h1lambda = h1r - h1;
        const float h0lambda = static_cast<float>(1) - h1lambda;
        //
        float w1r;
        area_pixel_compute_source_index(w1r, rwidth, w2, align_corners, /*cubic=*/false);
        const int w1 = w1r;
        const int w1p = (w1 < width1 - 1) ? 1 : 0;
        const float w1lambda = w1r - w1;
        const float w0lambda = static_cast<float>(1) - w1lambda;
        //
        for (int n = 0; n < batchsize; n++)
        {
            for (int c = 0; c < channels; ++c)
            {
                const float val = 
                    h0lambda * 
                    (w0lambda * idata[n * in_batchsize_stride + c * in_channels_stride + h1 * width1 + w1] +
                     w1lambda * idata[n * in_batchsize_stride + c * in_channels_stride + h1 * width1 + (w1 + w1p)]) +
                    h1lambda *
                    (w0lambda * idata[n * in_batchsize_stride + c * in_channels_stride + (h1 + h1p) * width1 + w1] +
                     w1lambda * idata[n * in_batchsize_stride + c * in_channels_stride + (h1 + h1p) * width1 + (w1 + w1p)]);
                odata[n * out_batchsize_stride + c * out_channels_stride + h2 * width2 + w2] = val;
            }
        }
    }
}

float ResizeBilinearPlugin::area_pixel_compute_scale(int input_size,
                                                   int output_size)
{
    if(output_size > 1)
    {
        return _align_corners ? float(input_size - 1) / (output_size - 1) : float(input_size) / output_size;
    }
    else
    {
        return 0.0;
    }
}

int ResizeBilinearPlugin::enqueue(int batchSize,
                                 const void *const *inputs, void **outputs,
                                 void *workspace, hipStream_t stream)
{
    auto const& input_dims = this->getInputDims(0);
    switch( _ndims )
    {
        case 2:
            {
                const int channels = input_dims.d[0];
                const int input_height = input_dims.d[1];
                const int input_width = input_dims.d[2];
                const int output_height = _output_dims.d[1];
                const int output_width = _output_dims.d[2];
                int obatchstride = _output_dims.d[1] * _output_dims.d[2];
                int num_kernels = obatchstride;
                int num_threads = 512;
                int blocks = int((num_kernels + num_threads - 1) / num_threads);
                int grid = num_threads;
                float rheight = area_pixel_compute_scale(input_height, output_height);
                float rwidth  = area_pixel_compute_scale(input_width, output_width);

                resize_bilinear_kernel_2d<<<blocks, grid, 0, stream>>>(
                        num_kernels, batchSize, channels, input_height, input_width,
                        output_height, output_width, rheight, rwidth, _align_corners,
                        static_cast<float const*>( inputs[0]), static_cast<float*>(outputs[0]));
                return hipGetLastError() != hipSuccess;
            }
        default: return -1;
    }
}
